#include "hip/hip_runtime.h"
/*
 * The Smith-Waterman algorithm, is a dynamic programming algorithm were the DP matrices 
 * involved in the computation are calculated dynamically. There are 3 DP: M, X, and Y 
 * each contributing a score from one of the three directions an entry in the SW scoring 
 * matrix can obtain. With the SW algorithm we implement affine-gap penalty scoring, thus
 * working towards a local alignment algortihm with affine-gap penalty as in the seed 
 * extension stage of BWA-MEM sequencing algorithm. 
 */
 #include "stdio.h"
 #include "string.h"
 #include "stdlib.h"
 #include "stdint.h"
 #include "swalign.h"
 #include <iostream>
 #include "hip/hip_runtime.h"

/*const int for penalty*/
const int penalty = gap_open + gap_extn;    
#define size ((L+1)/8 +1)


__host__ __device__ void init_DP(int M[][L+1], int X[][L+1], int Y[][L+1]){
	M[0][0] = 0;
	X[0][0] = -1000;
	Y[0][0] = -1000;
	for(int i=1; i <L+1; i++){
		M[i][0] = 0;
		X[i][0] = -1000;   //Just a large negative number
		Y[i][0] = -1000;
	}

	for(int j=1; j< L+1; j++){
		M[0][j] = 0;
		X[0][j] = -1000;   //Just a large negative number
		Y[0][j] = -1000;
	}
}

__host__ __device__ void unpacking(uint32_t *s1, char *seq1_out){
    uint8_t c;
    int m=0;
    for(int i=0; s1[i] !=0; i++){
        for(int j=0; j<8; j++){
           c = (s1[i] >> 4*j) & 0x000F;
           switch(c){
            case 0x1: { seq1_out[m] = 'A';
                           m++;
                          }
                          break;
            case 0x7: { seq1_out[m] = 'G';
                           m++;
                          }
                          break;
            case 0x4: { seq1_out[m] = 'T';
                           m++;
                          }
                          break;
            case 0x3: { seq1_out[m] = 'C';
                           m++;
                          }
                          break;
            case 0xA: { seq1_out[m] = '\n';
                           m++;
                          }
                          break;
            case 0xE: { seq1_out[m] = '-';
                           m++;
                          }
                          break;
            case 0xF: { seq1_out[m] = '.';
                           m++;
                          }
                          break;
            }
        
        }
    }
}

__global__ void read_align(char *sq1, char *sq2, char *seq1_out, char *seq2_out){
    
   int seq_i;
   sw_entry Score_Matrix[L+1][L+1];
   int M[L+1][L+1], X[L+1][L+1], Y[L+1][L+1];  //DP matrices
   int A, B, S_I;
   uint32_t *s1_out, *s2_out;

   
   uint32_t  seq1[size], seq2[size];

   int index = blockIdx.x * blockDim.x +threadIdx.x;
   
   if(index < no_seq)
   {   
        seq_i = index * size;
        	    
        /*Start scoring*/
       
        init_DP(M, X, Y);
        /*data packing*/
	    int p, j=0;
	    seq1[0] = 0x0;
	    seq2[0] = 0x0;
	    for(int i=0; i<L+1; i++){
		p = i%8;    
		switch(sq1[i]){
			case 'A': seq1[j] |= (sq1[i] & 0x0F) << 4*p;
				  break;
			case 'G': seq1[j] |= (sq1[i] & 0x0F) << 4*p;
				  break;
			case 'T': seq1[j] |= (sq1[i] & 0x0F) << 4*p;
				  break;
			case 'C': seq1[j] |= (sq1[i] & 0x0F) << 4*p;
				  break;
			case '\n': seq1[j] |= (sq1[i] & 0x0F) << 4*p;
				   break;
			case '-': seq1[j] |= (sq1[i] & 0x0F) << 4*p;
				  break;
		}
		switch(sq2[i]){
			case 'A': seq2[j] |= (sq2[i] & 0x0F) << 4*p;
				  break;
			case 'G': seq2[j] |= (sq2[i] & 0x0F) << 4*p;
				  break;
			case 'T': seq2[j] |= (sq2[i] & 0x0F) << 4*p;
				  break;
			case 'C': seq2[j] |= (sq2[i] & 0x0F) << 4*p;
				  break;
			case '\n': seq2[j] |= (sq2[i] & 0x0F) << 4*p;
				   break;
			case '-': seq2[j] |= (sq2[i] & 0x0F) << 4*p;
				  break;
		}
		if(p==7){
		  ++j;
		  seq1[j] = 0;
		  seq2[j] = 0;
		} 
	    }
      
      /*Initializing score martix*/  
            Score_Matrix[0][0].value = 0;
            for(int j=1; j<L+1; j++){
              Score_Matrix[0][j].value = 0;
            }
            for(int i=1; i<L+1; i++){
              Score_Matrix[i][0].value = 0;
            }
       A = M[0][0];
       seq1_out[A] = 'Z';
	   
 /*Compute DP matrices */
 /*     
    int M_max =0, X_max, Y_max;
    int M_x, M_y, M_m;
    int match_score;
    int si, sj, count=0;
    int r, c;
    uint8_t e1, e2;


    for(int I = 1; I < L+1; I=I+8){
       for(int J = 1; J <L+1; J=J+8){
	    r = I/8 + seq_i;
            c = J/8 + seq_i;
            for(int i=0; i<8; i++){
                for(int j=0; j<8; j++){
                    e1 = (seq1[r]>>((i+1)*4)) & (0x000F);
                    e2 = (seq2[c]>>((j+1)*4)) & (0x000F);
                       if(e1 == e2)
                        match_score = match;
                       else
                        match_score = mismatch;
                           
                       M_m = M[I+i-1][J+j-1] + match_score;
                       M_x = X[I+i-1][J+j-1] + match_score;
                       M_y = Y[I+i-1][J+j-1] + match_score;

                        if(M_m >= M_x && M_m >= M_y && M_m > 0) 
                            M_max = M_m;
                        else if(M_x >= M_m && M_x >= M_y && M_x > 0)
                            M_max = M_x;
                             else if(M_y >= M_m && M_y >= M_x && M_y > 0)
                                 M_max = M_y;

                        M[I+i][J+j] =  M_max;
                         
                        Y_max = gap_extn + Y[I+i][J+j-1];
                        if(penalty + M[I+i][J+j-1] > Y_max)
                        Y_max = M[I+i][J+j-1] + penalty;

                        Y[I+i][J+j] = Y_max;

                        X_max = gap_extn + X[I+i-1][J+j];
                        if(penalty + M[I+i-1][J+j] > X_max)
                        X_max = M[I+i-1][J+j] + penalty;

                        X[I+i][J+j] = X_max;


                        if(X_max >= Y_max && X_max >= M_max){
                        Score_Matrix[I+i][J+j].value = X_max;
                        Score_Matrix[I+i][J+j].direction = x;
                        }
                        else if(Y_max >= X_max && Y_max >= M_max){
                            Score_Matrix[I+i][J+j].value = Y_max;
                            Score_Matrix[I+i][J+j].direction = y;
                             }
                         else if(M_max >= X_max && M_max >= Y_max){
                             Score_Matrix[I+i][J+j].value = M_max;
                             Score_Matrix[I+i][J+j].direction = m;
                              }
                }
            }
        } 
      }
   */             
        //A = Score_Matrix[0][0].value;
	//seq1_out[A] = 'Y';
/*Maximum Score in SW matrix*/
/*  
	sw_entry sw_max;
	int val;

	sw_max = Score_Matrix[0][0];
	for(int i=0; i < L+1; i++){
		for(int j=0; j < L+1; j++){
			val = Score_Matrix[i][j].value;
			if(val > sw_max.value){
				sw_max.value = val;
				A = i;
				B = j;
				if(i >= j)
				  S_I = i;
				else
				  S_I = j;
			}
		}
          }
	//A = Score_Matrix[0][0].value;
        //seq2_out[B] = 'W';
*/	
   /*Traceback function*/
/*    
     DP_dir SW_dir;
     char c1, c2; 
     
     for(int n = L; n >=0; --n){
        s1_out[n/8 + seq_i] = 0;
        s2_out[n/8 + seq_i] = 0;
	if(M[A][B]!=0 && n <= S_I){  
       		SW_dir = Score_Matrix[A][B].direction;   
    		if(SW_dir == m){
                c1 = (seq1[A/8 + seq_i] >>(A%8))& 0x000F;
    			c2 = (seq2[B/8 + seq_i] >>(B%8))& 0x000F;
    			A = A-1;
    			B = B-1;
    		} else if(SW_dir == x){
    		        c2 = 0xE; 
    		   	c1 = (seq1[A/8 + seq_i] >>(A%8))& 0x000F;
    		   	A = A-1;
    			}
    	       		else if(SW_dir == y){
    	       	      		c1 = 0xE;
    	       	      		c2 = (seq2[B/8 + seq_i] >>(B%8)) & 0x000F;
    	       	      		B = B-1;
    	            		}
		    s1_out[n/8 + seq_i] |= (c1 << (A%8));
	        s2_out[n/8 + seq_i] |= (c2 << (B%8));
       } 
	 else if(M[A][B] == 0  && n <=S_I){//((M[A][B] != 0 && n > S_I)  || (M[A][B] == 0 && n <= S_I)){
		s1_out[n/8 + seq_i] |= (0xF << (A%8));
	        s2_out[n + seq_i] |= (0xF << (A%8));
	     }else if(M[A][B] !=0 && n >S_I){
		s1_out[n/8 + seq_i] |= (0xF << (A%8));
	        s2_out[n/8 + seq_i] |= (0xF << (A%8));
	     }	
     
     }
     unpacking(s1_out, seq1_out);
     unpacking(s2_out, seq2_out);        
*/
    }
}


/*Main function*/
int main(int argc, char *argv[]){
    
    FILE *input1, *input2;
    FILE *output;
   /*Read in the two sequences to be aligned, one from refrence and another a query
    *short read, which are stored in a text file and store in arrays seq1[] and seq2[]
    */
    input1 = fopen("seq1_out.txt","rb");
	if (!input1) {
	  printf("Unable to open input file %s.\n", "seq1_out.txt");
	  fflush(stdout);
	  exit(-1);
	}	
	input2 = fopen("seq2_out.txt","rb");
	if (!input2) {
	  printf("Unable to open input file %s.\n", "seq2_out.txt");
	  fflush(stdout);
	  exit(-1);
	}

    output = fopen("align_out.txt","wb");
    
    char *seq1, *seq2;
    char *seq1_out, *seq2_out;
    char line[] = "Output seq 1:";
    char line1[] = "Output seq 2:";
    char head[] = "Sequence pair";
    int l_size = strlen(line);
    size_t  s_size = no_seq * (L+1) * sizeof(char) ;
   

    /*Dynamic memory allocation at Host*/
    seq1 = (char*) malloc(s_size);
    if (seq1 == NULL) fprintf(stderr, "Bad malloc on seq1\n");
    seq2 = (char*) malloc(s_size);
    if (seq2 == NULL) fprintf(stderr, "Bad malloc on seq2\n");
    seq1_out = (char*) malloc(s_size);
    if (seq1_out == NULL) fprintf(stderr, "Bad malloc on seq1_out\n");
    seq2_out = (char*) malloc(s_size);
    if (seq2_out == NULL) fprintf(stderr, "Bad malloc on seq2_out\n");
 /*  
   int r_L;         //Reduced (L+1) is used as 8 sequence elements are stored in 1 entry, 4 bits each
   if((L+1)%8 != 0)
     r_L = ((L+1)/8 + 1);
   else
     r_L = (L+1)/8;

   size_t s_gpu = no_seq * r_L * sizeof(uint32_t);
  */ 
    /*Allocate memory in Device*/
    char *seq1_d;
    hipMalloc(&seq1_d, s_size);
    char *seq2_d;
    hipMalloc(&seq2_d, s_size);
    char *seq1_out_d;
    hipMalloc(&seq1_out_d, s_size);
    char *seq2_out_d;
    hipMalloc(&seq2_out_d, s_size);

    /* Load data from textfile */
    seq1[0] = '-';
    seq2[0] = '-';
    fread(&seq1[1], sizeof(char), ((L+1)*(no_seq-1)+ L), input1);
    fread(&seq2[1], sizeof(char), ((L+1)*(no_seq-1)+ L), input2);
     
   // printf("First char of seq1:%c, seq2:%c, last char of seq1:%c, seq2:%c\n", seq1[1], seq2[1], seq1[L], seq2[L]);

    fclose(input1);
    fclose(input2);
    fflush(stdout);
    
    //printf("Strlen of seq1:%d, seq2:%d\n", strlen(seq1), strlen(seq2));
    /*Copy data from Host to Device*/
    hipMemcpy(seq1_d, seq1, s_size, hipMemcpyHostToDevice);
    hipMemcpy(seq2_d, seq2, s_size, hipMemcpyHostToDevice);
   
    /*Perform alignment at Device*/
    read_align<<<1,no_seq>>>(seq1_d, seq2_d, seq1_out_d, seq2_out_d);
  
    hipDeviceSynchronize();
   
    /*Copy output data from Device to Host*/
    hipMemcpy(seq1_out, seq1_out_d, s_size, hipMemcpyDeviceToHost);
    hipMemcpy(seq2_out, seq2_out_d, s_size, hipMemcpyDeviceToHost);
    hipMemcpy(seq1, seq1_d, s_size, hipMemcpyDeviceToHost);
    hipMemcpy(seq2, seq2_d, s_size, hipMemcpyDeviceToHost);
    //printf("Strlen of seq1_out:%d, seq2_out:%d\n",strlen(seq1_out), strlen(seq2_out));
    /* Write result to file */
    for(int m=0; m < no_seq; m++){
	fwrite(head, sizeof(char), strlen(head), output);
        fprintf(output, "%d\n", m);	
        fwrite(line, sizeof(char), strlen(line), output);
        //fwrite(&seq1[m*(L+1)], sizeof(char), L+1, output);
        //fprintf(output,"\n");
        fwrite(&seq1_out[m*(L+1)], sizeof(char), L+1, output);
        fprintf(output,"\n");
        fwrite(line1, sizeof(char), strlen(line1), output);
        //fwrite(&seq2[m*(L+1)], sizeof(char), L+1, output);
        //fprintf(output, "\n");
        fwrite(&seq2_out[m*(L+1)], sizeof(char), L+1, output);
        if(m != no_seq-1)
          fprintf(output,"\n");
    }

	fclose(output);

	printf("Output complete.\n");
	fflush(stdout);

    /*Free Device memory*/
    hipFree(seq1_d);
    hipFree(seq2_d);
    hipFree(seq1_out_d);
    hipFree(seq2_out_d);

    /*Free Host memory*/
    free(seq1);
    free(seq2);
    free(seq1_out);
    free(seq2_out);
}
