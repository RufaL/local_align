#include "hip/hip_runtime.h"
/*
 * The Smith-Waterman algorithm, is a dynamic programming algorithm were the DP matrices 
 * involved in the computation are calculated dynamically. There are 3 DP: M, X, and Y 
 * each contributing a score from one of the three directions an entry in the SW scoring 
 * matrix can obtain. With the SW algorithm we implement affine-gap penalty scoring, thus
 * working towards a local alignment algortihm with affine-gap penalty as in the seed 
 * extension stage of BWA-MEM sequencing algorithm. 
 */
 #include "stdio.h"
 #include "string.h"
 #include "stdlib.h"
 #include "stdint.h"
 #include "swalign.h"
 #include <iostream>
 #include "hip/hip_runtime.h"

/*const int for penalty*/
const int penalty = gap_open + gap_extn;    

__device__ void init_DP(int M[][L+1], int X[][L+1], int Y[][L+1]){
	M[0][0] = 0;
	X[0][0] = -1000;
	Y[0][0] = -1000;
	for(int i=1; i <L+1; i++){
		M[i][0] = 0;
		X[i][0] = -1000;   //Just a large negative number
		Y[i][0] = -1000;
	}

	for(int j=1; j< L+1; j++){
		M[0][j] = 0;
		X[0][j] = -1000;   //Just a large negative number
		Y[0][j] = -1000;
	}
}

__device__ void compute_DP(sw_entry SW_i_j, int seq1_i, int seq2_i, char *seq1, char *seq2, int M[][L+1], int X[][L+1], int Y[][L+1]){
    int M_max =0, X_max, Y_max;
    int match_score;
    //printf("BEFORE\n");
    //printf("Index i:%d, j:%d, seq1:%c, seq2:%c, score:%d, dir:%d\n",seq1_i, seq2_i, seq1[seq1_i], seq2[seq2_i], SW_i_j.value, SW_i_j.direction);
   if(seq1[seq1_i] == seq2[seq2_i])
   	match_score = match;
   else
   	match_score = mismatch;

	if(M[seq1_i-1][seq2_i-1] + match_score> M_max)
		M_max = M[seq1_i-1][seq2_i-1] + match_score;
	if(X[seq1_i-1][seq2_i-1] + match_score> M_max)
		M_max = X[seq1_i-1][seq2_i-1] + match_score;
	if(Y[seq1_i-1][seq2_i-1] + match_score> M_max)
		M_max = Y[seq1_i-1][seq2_i-1] + match_score;

	M[seq1_i][seq2_i] =  M_max;

    Y_max = gap_extn + Y[seq1_i][seq2_i-1];
    if(penalty + M[seq1_i][seq2_i-1] > Y_max)
    	Y_max = M[seq1_i][seq2_i-1] + penalty;

    Y[seq1_i][seq2_i] = Y_max;

    X_max = gap_extn + X[seq1_i-1][seq2_i];
    if(penalty + M[seq1_i-1][seq2_i] > X_max)
    	X_max = M[seq1_i-1][seq2_i] + penalty;

    X[seq1_i][seq2_i] = X_max;

    SW_i_j.value = M_max;
    SW_i_j.direction = m;
    if(SW_i_j.value < X_max){
    	SW_i_j.value = X_max;
    	SW_i_j.direction = x;
    }
    if(SW_i_j.value < Y_max){
    	SW_i_j.value = Y_max;
    	SW_i_j.direction = y;
    }
    //printf("AFTER\n");
    //printf("Index i:%d, j:%d, seq1:%c, seq2:%c, score:%d, dir:%d\n",seq1_i, seq2_i, seq1[seq1_i], seq2[seq2_i], SW_i_j.value, SW_i_j.direction);


}

//__device__ sw_entry sw_max;
//__device__ int idx_i, idx_j;
__device__ void traceback(sw_entry SW[][L+1], int M[][L+1], char *seq1, char *seq2, char *seq1_out, char *seq2_out){
	sw_entry sw_max;
	int idx_i, idx_j;

	sw_max = SW[0][0];
	for(int i=0; i < L+1; i++){
		for(int j=0; j < L+1; j++){
			if(SW[i][j].value > sw_max.value){
				sw_max = SW[i][j];
				idx_i = i;
				idx_j = j;
			}
		}
          }
    //printf("Highest score index i:%d, j:%d and score:%d\n",idx_i, idx_j, sw_max.value);
    int I = idx_i, J = idx_j;
    int s_idx;
    if(idx_i > idx_j)
    	s_idx = idx_i;
    else
    	s_idx = idx_j;
    seq1_out[s_idx+1] ='\0';
    seq2_out[s_idx+1] ='\0';

    while(M[I][J]){
        //printf("**Index I:%d, J:%d, s_idx:%d char in seq1:%c, seq2:%c\n", I, J, s_idx, seq1[I], seq2[J]);
    	if(SW[I][J].direction == m){
                seq1_out[s_idx] = seq1[I];
    		seq2_out[s_idx] = seq2[J];
    		I = I-1;
    		J = J-1;
    	} else if(SW[I][J].direction == x){
    		     seq2_out[s_idx] = '-';
    		     seq1_out[s_idx] = seq1[I];
    		     I = I-1;
    		   }
    	       else {
    	       	 seq1_out[s_idx] = '-';
    	       	 seq2_out[s_idx] = seq2[J];
    	       	 J = J-1;
    	       }
      //printf("Score of M: %d\n", M[I][J]);
      //printf("Index I:%d, J:%d, char in seq1_out:%c, seq2_out:%c\n", I, J, seq1_out[s_idx], seq2_out[s_idx]);
      --s_idx;
    }

    while(s_idx > 0){
    seq1_out[s_idx] = '*';
    seq2_out[s_idx] = '*';
    --s_idx;
    }

}

__global__ void read_align(char *seq1, char *seq2, char *seq1_out, char *seq2_out){
    
   int seq_i;
   sw_entry Score_Matrix[L+1][L+1];
   int M[L+1][L+1], X[L+1][L+1], Y[L+1][L+1];  //DP matrices

   int index = blockIdx.x * blockDim.x +threadIdx.x;

   if(index < no_seq)
   {   
        seq_i = index * (L+1);
        seq1[seq_i] = '-';
        seq2[seq_i] = '-';
        seq1_out[seq_i] = '$';
        seq2_out[seq_i] = '$';
        /*Start scoring*/
        
        init_DP(M, X, Y);
      
            Score_Matrix[0][0].value = 0;
            for(int j=1; j<L+1; j++){
              Score_Matrix[0][j].value = 0;
            }
            for(int i=1; i<L+1; i++){
              Score_Matrix[i][0].value = 0;
            }

        for(int i=1; i<L+1; i++){
            for(int j=1; j<L+1; j++){
                compute_DP(Score_Matrix[i][j], i,j, &seq1[seq_i], &seq2[seq_i], M, X, Y);
            }
        }

        traceback(Score_Matrix, M, &seq1[seq_i], &seq2[seq_i], &seq1_out[seq_i], &seq2_out[seq_i]);
              

    }
}


/*Main function*/
int main(int argc, char *argv[]){
    
    FILE *input1, *input2;
    FILE *output;
   /*Read in the two sequences to be aligned, one from refrence and another a query
    *short read, which are stored in a text file and store in arrays seq1[] and seq2[]
    */
    //sprintf(buff1,argv[1]);
    //sprintf(buff2,argv[2]);
    input1 = fopen("seq1_out.txt","rb");//input1 = fopen(argv[1],"rb");
	if (!input1) {
	  printf("Unable to open input file %s.\n", "seq1_out.txt");//argv[1]);
	  fflush(stdout);
	  exit(-1);
	}	
	input2 = fopen("seq2_out.txt","rb");//input2 = fopen(argv[2],"rb");
	if (!input2) {
	  printf("Unable to open input file %s.\n", "seq2_out.txt");//argv[2]);
	  fflush(stdout);
	  exit(-1);
	}

    output = fopen("align_out.txt","wb");
    
    char *seq1, *seq2;
    char *seq1_out, *seq2_out;
    char line[] = "Output seq 1:";
    char line1[] = "Output seq 2:";
    int l_size = strlen(line);
    size_t  s_size = no_seq * (L+1) * sizeof(char) ;
   

    /*Dynamic memory allocation at Host*/
    seq1 = (char*) malloc(s_size);
    if (seq1 == NULL) fprintf(stderr, "Bad malloc on seq1\n");
    seq2 = (char*) malloc(s_size);
    if (seq2 == NULL) fprintf(stderr, "Bad malloc on seq2\n");
    seq1_out = (char*) malloc(s_size);
    if (seq1_out == NULL) fprintf(stderr, "Bad malloc on seq1_out\n");
    seq2_out = (char*) malloc(s_size);
    if (seq2_out == NULL) fprintf(stderr, "Bad malloc on seq2_out\n");
   
    /*Allocate memory in Device*/
    char *seq1_d;
    hipMalloc(&seq1_d, s_size);
    char *seq2_d;
    hipMalloc(&seq2_d, s_size);
    char *seq1_out_d;
    hipMalloc(&seq1_out_d, s_size);
    char *seq2_out_d;
    hipMalloc(&seq2_out_d, s_size);

    /* Load data from textfile */
    seq1[0] = '-';
    seq2[0] = '-';
    fread(&seq1[1], sizeof(char), ((L+1)*(no_seq-1)+ L), input1);
    fread(&seq2[1], sizeof(char), ((L+1)*(no_seq-1)+ L), input2);
     
    printf("First char of seq1:%c, seq2:%c, last char of seq1:%c, seq2:%c\n", seq1[1], seq2[1], seq1[L], seq2[L]);

    fclose(input1);
    fclose(input2);
    fflush(stdout);
    
    printf("Strlen of seq1:%d, seq2:%d\n", strlen(seq1), strlen(seq2));
    /*Copy data from Host to Device*/
    hipMemcpy(seq1_d, seq1, s_size, hipMemcpyHostToDevice);
    hipMemcpy(seq2_d, seq2, s_size, hipMemcpyHostToDevice);
   
    /*Perform alignment at Device*/
    read_align<<<1,1>>>(seq1_d, seq2_d, seq1_out_d, seq2_out_d);
  
    hipDeviceSynchronize();
   
    /*Copy output data from Device to Host*/
    hipMemcpy(seq1_out, seq1_out_d, s_size, hipMemcpyDeviceToHost);
    hipMemcpy(seq2_out, seq2_out_d, s_size, hipMemcpyDeviceToHost);
    printf("Strlen of seq1_out:%d, seq2_out:%d\n",strlen(seq1_out), strlen(seq2_out));
    /* Write result to file */
    for(int m=0; m < no_seq; m++){
        fwrite(line, sizeof(char), strlen(line), output);
        fwrite(&seq1[m*(L+1)], sizeof(char), L+1, output);
        fprintf(output,"\n");
        fwrite(&seq1_out[m*(L+1)], sizeof(char), L+1, output);
        fprintf(output,"\n");
        fwrite(line1, sizeof(char), strlen(line1), output);
        fwrite(&seq2[m*(L+1)], sizeof(char), L+1, output);
        fprintf(output, "\n");
        fwrite(&seq2_out[m*(L+1)], sizeof(char), L+1, output);
        if(m != no_seq-1)
          fprintf(output,"\n");
    }

	fclose(output);

	printf("Output complete.\n");
	fflush(stdout);

    /*Free Device memory*/
    hipFree(seq1_d);
    hipFree(seq2_d);
    hipFree(seq1_out_d);
    hipFree(seq2_out_d);

    /*Free Host memory*/
    free(seq1);
    free(seq2);
    free(seq1_out);
    free(seq2_out);

    return 0;
}
