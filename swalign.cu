#include "hip/hip_runtime.h"
/*
 * The Smith-Waterman algorithm, is a dynamic programming algorithm were the DP matrices 
 * involved in the computation are calculated dynamically. There are 3 DP: M, X, and Y 
 * each contributing a score from one of the three directions an entry in the SW scoring 
 * matrix can obtain. With the SW algorithm we implement affine-gap penalty scoring, thus
 * working towards a local alignment algortihm with affine-gap penalty as in the seed 
 * extension stage of BWA-MEM sequencing algorithm. 
 */
 #include "stdio.h"
 #include "string.h"
 #include "stdlib.h"
 #include "stdint.h"
 #include "swalign.h"
 #include <iostream>
 #include "hip/hip_runtime.h"

/*const int for penalty*/
const int penalty = gap_open + gap_extn;    

__host__ __device__ void init_DP(int16_t M[][L+1], int16_t X[][L+1], int16_t Y[][L+1]){
	M[0][0] = 0;
	X[0][0] = -1000;
	Y[0][0] = -1000;
	for(int i=1; i <2; i++){
		M[i][0] = 0;
		X[i][0] = -1000;   //Just a large negative number
		Y[i][0] = -1000;
	}

	for(int j=1; j< L+1; j++){
		M[0][j] = 0;
		X[0][j] = -1000;   //Just a large negative number
		Y[0][j] = -1000;
	}
}


__global__ void read_align(char *seq1, char *seq2, char *seq1_out, char *seq2_out){
    
   int seq_i;
   uint16_t Score_Matrix[L+1][L+1];
   uint8_t Dir[L][L/2];
   int16_t M[2][L+1], X[2][L+1], Y[2][L+1];  //DP matrices
   int A, B, S_I;

   int index = blockIdx.x * blockDim.x +threadIdx.x;
   
   if(index < no_seq)
   {   
        seq_i = index * (L+1);
        seq1[seq_i] = '-';
        seq2[seq_i] = '-';
        seq1_out[seq_i] = '$';
        seq2_out[seq_i] = '$';
        /*Start scoring*/
       
        init_DP(M, X, Y);

	Score_Matrix[0][0] = 0;
	for(int i=1; i <L+1; i++){
		Score_Matrix[i][0] = 0;
	        Score_Matrix[0][i] = 0;
	}
       //A = M[0][0];
       //seq1_out[A] = 'Z';
	   
 /*Compute DP matrices */
    int M_max =0, X_max, Y_max;
    int M_x, M_y, M_m;
    int match_score;
    int si, sj;
    int sw_max;
    Dir[0][0] = 0;
    int d_I=0, d_J=0, d_p=0;
   

    for(int I = 1; I < L+1; I++){
       for(int J = 1; J <L+1; J++){
			  si = I + seq_i;
			  sj = J + seq_i;
	   if(seq1[si] == seq2[sj])
		match_score = match;
	   else
		match_score = mismatch;
           
	   M_m = M[(I-1)%2][J-1] + match_score;
	   M_x  = X[(I-1)%2][J-1] + match_score;
	   M_y = Y[(I-1)%2][J-1] + match_score;

	        M_max =0;
		if(M_m >= M_x && M_m >= M_y && M_m > 0) 
			M_max = M_m;
		else if(M_x >= M_m && M_x >= M_y && M_x > 0)
			M_max = M_x;
		     else if(M_y >= M_m && M_y >= M_x && M_y > 0)
			     M_max = M_y;

		M[I%2][J] =  M_max;
         
	    Y_max = gap_extn + Y[I%2][J-1];
	    if(penalty + M[I%2][J-1] > Y_max)
		Y_max = M[I%2][J-1] + penalty;

	    Y[I%2][J] = Y_max;

	    X_max = gap_extn + X[(I-1)%2][J];
	    if(penalty + M[(I-1)%2][J] > X_max)
		X_max = M[(I-1)%2][J] + penalty;

	    X[I%2][J] = X_max;

            
	    if(X_max >= Y_max && X_max >= M_max){
	       Score_Matrix[I][J] = X_max;
	       Dir[d_I][d_J] |= (0x02 << 4*d_p);
	    }
	    else if(Y_max >= X_max && Y_max >= M_max){
		    Score_Matrix[I][J] = Y_max;
		    Dir[d_I][d_J] |= (0x03 << 4*d_p);
		 }
		 else if(M_max >= X_max && M_max >= Y_max){
			Score_Matrix[I][J] = M_max;
			Dir[d_I][d_J] |= (0x01 << 4*d_p);
		 }
           ++d_p;		 

           if(d_p == 2){
              d_p = 0;
	      ++d_J;
	      if(d_J == L/2){
		      d_J = 0;
		      ++d_I;
	      }
	      if(d_I < L && d_J < L/2)
	      	      Dir[d_I][d_J] = 0;
	   }

	   if(Score_Matrix[I][J] > sw_max){
		   A = I;
		   B = J;
		   sw_max = Score_Matrix[I][J];
	   }


	} 
      }
   	
      if(A >= B)
	      S_I = A;
      else
	      S_I = B;

        //A = Score_Matrix[0][0].value;
	//seq1_out[A] = 'Y';	
   /*Traceback function*/
    
     uint8_t SW_dir;
     char c1, c2; 
     int p_t;
     
     for(int n = L; n >=0; --n){
	if(Score_Matrix[A][B]!=0 && n <= S_I){ 
	        p_t = 1 - B%2;	
		if(A>=1 && B>=1){
		if(B%2 == 1)	
       		   SW_dir = (Dir[A-1][(B-1)/2] >> 4*p_t) & 0x0F;
		else
                   SW_dir = (Dir[A-1][B/2 - 1] >> 4*p_t) & 0x0F;

    		if(SW_dir == 0x01){
                	c1 = seq1[A + seq_i];
    			c2 = seq2[B + seq_i];
    			A = A-1;
    			B = B-1;
    		} else if(SW_dir == 0x02){
    		        c2 = '-'; 
    		   	c1 = seq1[A + seq_i];
    		   	A = A-1;
    			}
    	       		else if(SW_dir == 0x03){
    	       	      		c1 = '-';
    	       	      		c2 = seq2[B + seq_i];
    	       	      		B = B-1;
    	            		}
				
		seq1_out[n + seq_i] = c1;
	        seq2_out[n + seq_i] = c2;
		}
       } 
	 else if(Score_Matrix[A][B] == 0  && n <=S_I){//((M[A][B] != 0 && n > S_I)  || (M[A][B] == 0 && n <= S_I)){
		seq1_out[n + seq_i] = '.';
	        seq2_out[n + seq_i] = '.';
	     }else if(Score_Matrix[A][B] !=0 && n >S_I){
		seq1_out[n + seq_i] = '*';
	        seq2_out[n + seq_i] = '*';
	     }	
     
	 }
	 
             

    }
}


/*Main function*/
int main(int argc, char *argv[]){
    
    FILE *input1, *input2;
    FILE *output;
   /*Read in the two sequences to be aligned, one from refrence and another a query
    *short read, which are stored in a text file and store in arrays seq1[] and seq2[]
    */
    //sprintf(buff1,argv[1]);
    //sprintf(buff2,argv[2]);
    input1 = fopen("seq1_out.txt","rb");//input1 = fopen(argv[1],"rb");
	if (!input1) {
	  printf("Unable to open input file %s.\n", "seq1_out.txt");//argv[1]);
	  fflush(stdout);
	  exit(-1);
	}	
	input2 = fopen("seq2_out.txt","rb");//input2 = fopen(argv[2],"rb");
	if (!input2) {
	  printf("Unable to open input file %s.\n", "seq2_out.txt");//argv[2]);
	  fflush(stdout);
	  exit(-1);
	}

    output = fopen("align_out.txt","wb");
    
    char *seq1, *seq2;
    char *seq1_out, *seq2_out;
    char line[] = "Output seq 1:";
    char line1[] = "Output seq 2:";
    char head[] = "Sequence pair";
    int l_size = strlen(line);
    size_t  s_size = no_seq * (L+1) * sizeof(char) ;
    hipEvent_t start, stop;
    float milliseconds;
    hipEventCreate(&start);
    hipEventCreate(&stop);
   

    /*Dynamic memory allocation at Host*/
    seq1 = (char*) malloc(s_size);
    if (seq1 == NULL) fprintf(stderr, "Bad malloc on seq1\n");
    seq2 = (char*) malloc(s_size);
    if (seq2 == NULL) fprintf(stderr, "Bad malloc on seq2\n");
    seq1_out = (char*) malloc(s_size);
    if (seq1_out == NULL) fprintf(stderr, "Bad malloc on seq1_out\n");
    seq2_out = (char*) malloc(s_size);
    if (seq2_out == NULL) fprintf(stderr, "Bad malloc on seq2_out\n");
   
    /*Allocate memory in Device*/
    char *seq1_d;
    hipMalloc(&seq1_d, s_size);
    char *seq2_d;
    hipMalloc(&seq2_d, s_size);
    char *seq1_out_d;
    hipMalloc(&seq1_out_d, s_size);
    char *seq2_out_d;
    hipMalloc(&seq2_out_d, s_size);

    /* Load data from textfile */
    seq1[0] = '-';
    seq2[0] = '-';
    fread(&seq1[1], sizeof(char), ((L+1)*(no_seq-1)+ L), input1);
    fread(&seq2[1], sizeof(char), ((L+1)*(no_seq-1)+ L), input2);
     
   // printf("First char of seq1:%c, seq2:%c, last char of seq1:%c, seq2:%c\n", seq1[1], seq2[1], seq1[L], seq2[L]);

    fclose(input1);
    fclose(input2);
    fflush(stdout);
    
    hipEventRecord(start);
    /*Copy data from Host to Device*/
    hipMemcpy(seq1_d, seq1, s_size, hipMemcpyHostToDevice);
    hipMemcpy(seq2_d, seq2, s_size, hipMemcpyHostToDevice);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Time in H to D:%4.4f\n", milliseconds);
  
    hipEventRecord(start);
    /*Perform alignment at Device*/
    read_align<<<1, (no_seq)>>>(seq1_d, seq2_d, seq1_out_d, seq2_out_d);
    hipEventRecord(stop);
    hipDeviceSynchronize();
    hipEventSynchronize(stop);
    milliseconds =0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Time in kernel:%4.4f\n", milliseconds);
   
    hipEventRecord(start);
    /*Copy output data from Device to Host*/
    hipMemcpy(seq1_out, seq1_out_d, s_size, hipMemcpyDeviceToHost);
    hipMemcpy(seq2_out, seq2_out_d, s_size, hipMemcpyDeviceToHost);
    //hipMemcpy(seq1, seq1_d, s_size, hipMemcpyDeviceToHost);
    //hipMemcpy(seq2, seq2_d, s_size, hipMemcpyDeviceToHost);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    milliseconds =0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Time in D to H:%4.4f\n",milliseconds);

    /* Write result to file */
    for(int m=0; m < no_seq; m++){
	fwrite(head, sizeof(char), strlen(head), output);
        fprintf(output, "%d\n", m);	
        fwrite(line, sizeof(char), strlen(line), output);
        //fwrite(&seq1[m*(L+1)], sizeof(char), L+1, output);
        //fprintf(output,"\n");
        fwrite(&seq1_out[m*(L+1)], sizeof(char), L+1, output);
        fprintf(output,"\n");
        fwrite(line1, sizeof(char), strlen(line1), output);
        //fwrite(&seq2[m*(L+1)], sizeof(char), L+1, output);
        //fprintf(output, "\n");
        fwrite(&seq2_out[m*(L+1)], sizeof(char), L+1, output);
        if(m != no_seq-1)
          fprintf(output,"\n");
    }

	fclose(output);

	printf("Output complete.\n");
	fflush(stdout);

        hipError_t err = hipGetLastError();
	if (err != hipSuccess) 
          printf("Error: %s\n", hipGetErrorString(err));

    /*Free Device memory*/
    hipFree(seq1_d);
    hipFree(seq2_d);
    hipFree(seq1_out_d);
    hipFree(seq2_out_d);

    /*Free Host memory*/
    free(seq1);
    free(seq2);
    free(seq1_out);
    free(seq2_out);
}
